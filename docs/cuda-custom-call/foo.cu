#include "hip/hip_runtime.h"
#include <xla/ffi/api/c_api.h>
#include <xla/ffi/api/api.h>
#include <xla/ffi/api/ffi.h>

// c = a * (b+1)
// This strawman operation works well for demo purposes because:
// 1. it's simple enough to be quickly understood,
// 2. it's complex enough to require intermediate outputs in grad computation,
//    like many operations in practice do, and
// 3. it does not have a built-in implementation in JAX.
__global__ void FooKernel(const float *a, const float *b, float *c, size_t n) {
  size_t tid = blockIdx.x * blockDim.x + threadIdx.x;
  const size_t grid_stride = blockDim.x * gridDim.x;
  for (size_t i = tid; i < n; i += grid_stride)
    c[i] = a[i] * (b[i] + 1.0f);
}

namespace ffi = xla::ffi;

// XLA FFI binding wrapper that launches the kernel
extern "C" XLA_FFI_Error *Foo(XLA_FFI_CallFrame *call_frame) {
  static const auto *kImpl =
      ffi::Ffi::Bind()
          .Ctx<ffi::PlatformStream<hipStream_t>>()
          .Arg<ffi::Buffer<ffi::DataType::F32>>()
          .Arg<ffi::Buffer<ffi::DataType::F32>>()
          .Ret<ffi::Buffer<ffi::DataType::F32>>()
          .Attr<size_t>("n")
          .To([](hipStream_t stream,
                 ffi::Buffer<ffi::DataType::F32> a,
                 ffi::Buffer<ffi::DataType::F32> b,
                 ffi::Result<ffi::Buffer<ffi::DataType::F32>> c,
                 size_t n) -> ffi::Error {
            // Host function wrapper that launches the kernel with hardcoded
            // grid/block size. Note, it uses types from XLA FFI. The return
            // type must be ffi::Error. Buffer type provides buffer dimensions,
            // so the "n" argument here is not strictly necessary, but it allows
            // us to demonstrate the use of attributes (.Attr in the FFI handler
            // definition above).
            const int block_dim = 128;
            const int grid_dim = 1;
            // Note how we access regular Buffer data vs Result Buffer data:
            FooKernel<<<grid_dim, block_dim, /*shared_mem=*/0, stream>>>(
                a.data, b.data, c->data, n);
            // Check for launch time errors. Note that this function may also
            // return error codes from previous, asynchronous launches. This
            // means that an error status returned here could have been caused
            // by a different kernel previously launched by XLA.
            hipError_t last_error = hipGetLastError();
            if (last_error != hipSuccess) {
              return ffi::Error(XLA_FFI_Error_Code_INTERNAL,
                                std::string("CUDA error: ") +
                                hipGetErrorString(last_error));
            }
            return ffi::Error::Success();
      }).release();
  return kImpl->Call(call_frame);
}
